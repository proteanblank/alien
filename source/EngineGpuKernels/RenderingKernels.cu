#include "hip/hip_runtime.h"
﻿#include "RenderingKernels.cuh"

namespace
{
    __device__ __inline__ void drawPixel(uint64_t* imageData, unsigned int index, float3 const& color)
    {
        imageData[index] = toUInt64(color.y * 225.0f) << 16 | toUInt64(color.x * 225.0f) << 0 | toUInt64(color.z * 225.0f) << 32;
    }

    __device__ __inline__ void drawAddingPixel(uint64_t* imageData, unsigned int index, float3 const& colorToAdd)
    {
        uint64_t rawColorToAdd = toUInt64(colorToAdd.y * 255.0f) << 16 | toUInt64(colorToAdd.x * 255.0f) << 0 | toUInt64(colorToAdd.z * 255.0f) << 32;
        alienAtomicAdd(&imageData[index], rawColorToAdd);
    }

    __device__ __inline__ float3 colorToFloat3(unsigned int value)
    {
        return float3{toFloat(value & 0xff) / 255, toFloat((value >> 8) & 0xff) / 255, toFloat((value >> 16) & 0xff) / 255};
    }

    __device__ __inline__ float3 mix(float3 const& a, float3 const& b, float factor)
    {
        return float3{a.x * factor + b.x * (1 - factor), a.y * factor + b.y * (1 - factor), a.z * factor + b.z * (1 - factor)};
    }

    __device__ __inline__ float3 mix(float3 const& a, float3 const& b, float3 const& c, float factor1, float factor2)
    {
        float weight1 = factor1 * factor2;
        float weight2 = 1 - factor1;
        float weight3 = 1 - factor2;
        float sum = weight1 + weight2 + weight3;
        weight1 /= sum;
        weight2 /= sum;
        weight3 /= sum;
        return float3{
            a.x * weight1 + b.x * weight2 + c.x * weight3, a.y * weight1 + b.y * weight2 + c.y * weight3, a.z * weight1 + b.z * weight2 + c.z * weight3};
    }

    __device__ __inline__ float2 mapUniversePosToVectorImagePos(float2 const& rectUpperLeft, float2 const& pos, float zoom)
    {
        return float2{(pos.x - rectUpperLeft.x) * zoom, (pos.y - rectUpperLeft.y) * zoom};
    }

    __device__ __inline__ float3 calcColor(Cell* cell, int selected)
    {
        uint32_t cellColor;
        switch (cell->metadata.color % 7) {
        case 0: {
            cellColor = Const::IndividualCellColor1;
            break;
        }
        case 1: {
            cellColor = Const::IndividualCellColor2;
            break;
        }
        case 2: {
            cellColor = Const::IndividualCellColor3;
            break;
        }
        case 3: {
            cellColor = Const::IndividualCellColor4;
            break;
        }
        case 4: {
            cellColor = Const::IndividualCellColor5;
            break;
        }
        case 5: {
            cellColor = Const::IndividualCellColor6;
            break;
        }
        case 6: {
            cellColor = Const::IndividualCellColor7;
            break;
        }
        }

        float factor = min(300.0f, cell->energy) / 320.0f;
        if (1 == selected) {
            factor *= 2.5f;
        }
        if (2 == selected) {
            factor *= 1.75f;
        }

        return {
            toFloat((cellColor >> 16) & 0xff) / 256.0f * factor,
            toFloat((cellColor >> 8) & 0xff) / 256.0f * factor,
            toFloat(cellColor & 0xff) / 256.0f * factor};
    }

    __device__ __inline__ float3 calcColor(Particle* particle, bool selected)
    {
        auto intensity = max(min((toInt(particle->energy) + 10) * 5, 150), 20) / 266.0f;
        if (selected) {
            intensity *= 2.5f;
        }

        return {intensity, 0, 0.08f};
    }

    __device__ __inline__ float3 calcColor(bool selected) { return selected ? float3{0.75f, 0.75f, 0.75f} : float3{0.5f, 0.5f, 0.5f}; }

    __device__ __inline__ void drawDot(uint64_t* imageData, int2 const& imageSize, float2 const& pos, float3 const& colorToAdd)
    {
        int2 intPos{toInt(pos.x), toInt(pos.y)};
        if (intPos.x >= 1 && intPos.x < imageSize.x - 1 && intPos.y >= 1 && intPos.y < imageSize.y - 1) {

            float2 posFrac{pos.x - intPos.x, pos.y - intPos.y};
            unsigned int index = intPos.x + intPos.y * imageSize.x;

            float3 colorToAdd1 = colorToAdd * (1.0f - posFrac.x) * (1.0f - posFrac.y);
            drawAddingPixel(imageData, index, colorToAdd1);

            float3 colorToAdd2 = colorToAdd * posFrac.x * (1.0f - posFrac.y);
            drawAddingPixel(imageData, index, colorToAdd2);

            float3 colorToAdd3 = colorToAdd * (1.0f - posFrac.x) * posFrac.y;
            drawAddingPixel(imageData, index + imageSize.x, colorToAdd3);

            float3 colorToAdd4 = colorToAdd * posFrac.x * posFrac.y;
            drawAddingPixel(imageData, index + imageSize.x + 1, colorToAdd4);
        }
    }

    __device__ __inline__ void drawCircle(uint64_t* imageData, int2 const& imageSize, float2 pos, float3 color, float radius, bool inverted = false)
    {
        if (radius > 1.5 - FP_PRECISION) {
            auto radiusSquared = radius * radius;
            for (float x = -radius; x <= radius; x += 1.0f) {
                for (float y = -radius; y <= radius; y += 1.0f) {
                    auto rSquared = x * x + y * y;
                    if (rSquared <= radiusSquared) {
                        auto factor = inverted ? (rSquared / radiusSquared) * 2 : (1.0f - rSquared / radiusSquared) * 2;
                        drawDot(imageData, imageSize, pos + float2{x, y}, color * min(factor, 1.0f));
                    }
                }
            }
        } else {
            color = color * radius * 2;
            drawDot(imageData, imageSize, pos, color);
            color = color * 0.3f;
            drawDot(imageData, imageSize, pos + float2{1, 0}, color);
            drawDot(imageData, imageSize, pos + float2{-1, 0}, color);
            drawDot(imageData, imageSize, pos + float2{0, 1}, color);
            drawDot(imageData, imageSize, pos + float2{0, -1}, color);
        }
    }

    __device__ __inline__ void
    drawLine(float2 const& start, float2 const& end, float3 const& color, uint64_t* imageData, int2 imageSize, float pixelDistance = 1.5f)
    {
        float dist = Math::length(end - start);
        float2 const v = {static_cast<float>(end.x - start.x) / dist * pixelDistance, static_cast<float>(end.y - start.y) / dist * pixelDistance};
        float2 pos = start;

        for (float d = 0; d <= dist; d += pixelDistance) {
            drawDot(imageData, imageSize, pos, color);
            pos = pos + v;
        }
    }
}

/************************************************************************/
/* Main      															*/
/************************************************************************/
__global__ void cudaDrawBackground(uint64_t* imageData, int2 imageSize, int2 worldSize, float zoom, float2 rectUpperLeft, float2 rectLowerRight)
{
    int2 outsideRectUpperLeft{-min(toInt(rectUpperLeft.x * zoom), 0), -min(toInt(rectUpperLeft.y * zoom), 0)};
    int2 outsideRectLowerRight{
        imageSize.x - max(toInt((rectLowerRight.x - worldSize.x) * zoom), 0), imageSize.y - max(toInt((rectLowerRight.y - worldSize.y) * zoom), 0)};

    MapInfo map;
    map.init(worldSize);
    auto spaceColor = colorToFloat3(Const::SpaceColor);
    auto spotColor1 = colorToFloat3(cudaSimulationParametersSpots.spots[0].color);
    auto spotColor2 = colorToFloat3(cudaSimulationParametersSpots.spots[1].color);

    auto const block = calcPartition(imageSize.x * imageSize.y, threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);
    for (int index = block.startIndex; index <= block.endIndex; ++index) {
        auto x = index % imageSize.x;
        auto y = index / imageSize.x;
        if (x < outsideRectUpperLeft.x || y < outsideRectUpperLeft.y || x >= outsideRectLowerRight.x || y >= outsideRectLowerRight.y) {
            imageData[index] = 0;
        } else {
            if (0 == cudaSimulationParametersSpots.numSpots) {
                drawPixel(imageData, index, spaceColor);
            }
            if (1 == cudaSimulationParametersSpots.numSpots) {
                float2 worldPos = {toFloat(x) / zoom + rectUpperLeft.x, toFloat(y) / zoom + rectUpperLeft.y};
                auto distance = map.mapDistance(worldPos, {cudaSimulationParametersSpots.spots[0].posX, cudaSimulationParametersSpots.spots[0].posY});
                auto coreRadius = cudaSimulationParametersSpots.spots[0].coreRadius;
                auto fadeoutRadius = cudaSimulationParametersSpots.spots[0].fadeoutRadius + 1;
                auto factor = distance < coreRadius ? 0.0f : min(1.0f, (distance - coreRadius) / fadeoutRadius);
                auto resultingColor = mix(spaceColor, spotColor1, factor);
                drawPixel(imageData, index, resultingColor);
            }
            if (2 == cudaSimulationParametersSpots.numSpots) {
                float2 worldPos = {toFloat(x) / zoom + rectUpperLeft.x, toFloat(y) / zoom + rectUpperLeft.y};
                auto distance1 = map.mapDistance(worldPos, {cudaSimulationParametersSpots.spots[0].posX, cudaSimulationParametersSpots.spots[0].posY});
                auto distance2 = map.mapDistance(worldPos, {cudaSimulationParametersSpots.spots[1].posX, cudaSimulationParametersSpots.spots[1].posY});

                auto coreRadius1 = cudaSimulationParametersSpots.spots[0].coreRadius;
                auto fadeoutRadius1 = cudaSimulationParametersSpots.spots[0].fadeoutRadius + 1;
                auto factor1 = distance1 < coreRadius1 ? 0.0f : min(1.0f, (distance1 - coreRadius1) / fadeoutRadius1);
                auto coreRadius2 = cudaSimulationParametersSpots.spots[1].coreRadius;
                auto fadeoutRadius2 = cudaSimulationParametersSpots.spots[1].fadeoutRadius + 1;
                auto factor2 = distance2 < coreRadius2 ? 0.0f : min(1.0f, (distance2 - coreRadius2) / fadeoutRadius2);

                auto resultingColor = mix(spaceColor, spotColor1, spotColor2, factor1, factor2);
                drawPixel(imageData, index, resultingColor);
            }
        }
    }
}

__global__ void cudaDrawCells(int2 universeSize, float2 rectUpperLeft, float2 rectLowerRight, Array<Cell*> cells, uint64_t* imageData, int2 imageSize, float zoom)
{
    auto const partition = calcPartition(cells.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    MapInfo map;
    map.init(universeSize);

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        auto const& cell = cells.at(index);

        auto cellPos = cell->absPos;
        map.mapPosCorrection(cellPos);
        if (isContainedInRect(rectUpperLeft, rectLowerRight, cellPos)) {
            auto cellImagePos = mapUniversePosToVectorImagePos(rectUpperLeft, cellPos, zoom);
            auto color = calcColor(cell, cell->selected);
            auto radius = 1 == cell->selected ? zoom / 2 : zoom / 3;
            drawCircle(imageData, imageSize, cellImagePos, color, radius, true);
            color = color * min((zoom - 1.0f) / 3, 1.0f);

            //draw connection lines
            if (zoom >= 1.0f) {
                for (int i = 0; i < cell->numConnections; ++i) {
                    auto const otherCell = cell->connections[i].cell;
                    auto const otherCellPos = otherCell->absPos;
                    auto topologyCorrection = map.correctionIncrement(cellPos, otherCellPos);
                    if (Math::lengthSquared(topologyCorrection) < FP_PRECISION) {
                        auto const otherCellImagePos = mapUniversePosToVectorImagePos(rectUpperLeft, otherCellPos, zoom);
                        drawLine(cellImagePos, otherCellImagePos, color, imageData, imageSize);
                    }
                }
            }

            //draw arrows
            if (zoom >= 15.0f) {
                for (int i = 0; i < cell->numConnections; ++i) {
                    auto const otherCell = cell->connections[i].cell;
                    auto const otherCellPos = otherCell->absPos;
                    auto topologyCorrection = map.correctionIncrement(cellPos, otherCellPos);
                    if (Math::lengthSquared(topologyCorrection) > FP_PRECISION) {
                        continue;
                    }
                    if ((cell->branchNumber + 1 - otherCell->branchNumber) % cudaSimulationParameters.cellMaxTokenBranchNumber == 0) {
                        auto const arrowEnd =
                            mapUniversePosToVectorImagePos(rectUpperLeft, otherCellPos + Math::normalized(cellPos - otherCellPos) / 3, zoom);
                        auto direction = Math::normalized(arrowEnd - cellImagePos);
                        {
                            float2 arrowPartStart = {-direction.x + direction.y, -direction.x - direction.y};
                            arrowPartStart = arrowPartStart * zoom / 6 + arrowEnd;
                            drawLine(arrowPartStart, arrowEnd, color, imageData, imageSize, 0.7f);
                        }
                        {
                            float2 arrowPartStart = {-direction.x - direction.y, direction.x - direction.y};
                            arrowPartStart = arrowPartStart * zoom / 6 + arrowEnd;
                            drawLine(arrowPartStart, arrowEnd, color, imageData, imageSize, 0.7f);
                        }
                    }
                    if ((cell->branchNumber - 1 - otherCell->branchNumber) % cudaSimulationParameters.cellMaxTokenBranchNumber == 0) {
                        auto const arrowEnd = mapUniversePosToVectorImagePos(rectUpperLeft, cellPos + Math::normalized(otherCellPos - cellPos) / 3, zoom);
                        auto const otherCellImagePos = mapUniversePosToVectorImagePos(rectUpperLeft, otherCellPos, zoom);
                        auto direction = Math::normalized(arrowEnd - otherCellImagePos);
                        {
                            float2 arrowPartStart = {-direction.x + direction.y, -direction.x - direction.y};
                            arrowPartStart = arrowPartStart * zoom / 6 + arrowEnd;
                            drawLine(arrowPartStart, arrowEnd, color, imageData, imageSize, 0.7f);
                        }
                        {
                            float2 arrowPartStart = {-direction.x - direction.y, direction.x - direction.y};
                            arrowPartStart = arrowPartStart * zoom / 6 + arrowEnd;
                            drawLine(arrowPartStart, arrowEnd, color, imageData, imageSize, 0.7f);
                        }
                    }
                }
            }
        }
    }
}

__global__ void
cudaDrawTokens(int2 universeSize, float2 rectUpperLeft, float2 rectLowerRight, Array<Token*> tokens, uint64_t* imageData, int2 imageSize, float zoom)
{
    MapInfo map;
    map.init(universeSize);

    auto partition = calcPartition(tokens.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);
    for (auto tokenIndex = partition.startIndex; tokenIndex <= partition.endIndex; ++tokenIndex) {
        auto const& token = tokens.at(tokenIndex);
        auto const& cell = token->cell;

        auto cellPos = cell->absPos;
        map.mapPosCorrection(cellPos);
        auto const cellImagePos = mapUniversePosToVectorImagePos(rectUpperLeft, cellPos, zoom);
        if (isContainedInRect({0, 0}, imageSize, cellImagePos)) {
            auto const color = calcColor(false);
            drawCircle(imageData, imageSize, cellImagePos, color, zoom / 2);
        }
    }
}

__global__ void
cudaDrawParticles(int2 universeSize, float2 rectUpperLeft, float2 rectLowerRight, Array<Particle*> particles, uint64_t* imageData, int2 imageSize, float zoom)
{
    auto const particleBlock = calcPartition(particles.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    for (int index = particleBlock.startIndex; index <= particleBlock.endIndex; ++index) {
        auto const& particle = particles.at(index);

        auto const particleImagePos = mapUniversePosToVectorImagePos(rectUpperLeft, particle->absPos, zoom);
        if (isContainedInRect({0, 0}, imageSize, particleImagePos)) {
            auto const color = calcColor(particle, 0 != particle->selected);
            auto radius = 1 == particle->selected ? zoom / 2 : zoom / 3;
            drawCircle(imageData, imageSize, particleImagePos, color, radius);
        }
    }
}

__global__ void cudaDrawFlowCenters(uint64_t* targetImage, float2 rectUpperLeft, int2 imageSize, float zoom)
{
    if (cudaFlowFieldSettings.active) {
        for (int i = 0; i < cudaFlowFieldSettings.numCenters; ++i) {
            auto const& radialFlowData = cudaFlowFieldSettings.centers[i];
            int screenPosX = toInt(radialFlowData.posX * zoom) - rectUpperLeft.x * zoom;
            int screenPosY = toInt(radialFlowData.posY * zoom) - rectUpperLeft.y * zoom;
            auto drawX = screenPosX;
            auto drawY = screenPosY;
            if (0 <= drawX && drawX < imageSize.x && 0 <= drawY && drawY < imageSize.y) {
                int index = drawX + drawY * imageSize.x;
                targetImage[index] = 0xffff00000000;
            }
        }
    }
}
