#include "hip/hip_runtime.h"
#include "CudaSimulationAdapter.cuh"

#include <functional>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <>
#include <cuda/hip/hip_runtime_api.h>

#include "Base/Exceptions.h"
#include "EngineInterface/InspectedEntityIds.h"
#include "EngineInterface/SimulationParameters.h"
#include "EngineInterface/GpuSettings.h"

#include "Base/LoggingService.h"
#include "DataAccessKernels.cuh"
#include "AccessTOs.cuh"
#include "Base.cuh"
#include "GarbageCollectorKernels.cuh"
#include "ConstantMemory.cuh"
#include "CudaMemoryManager.cuh"
#include "CudaMonitorData.cuh"
#include "Entities.cuh"
#include "Map.cuh"
#include "MonitorKernels.cuh"
#include "EditKernels.cuh"
#include "RenderingKernels.cuh"
#include "SimulationData.cuh"
#include "SimulationKernelsLauncher.cuh"
#include "DataAccessKernelsLauncher.cuh"
#include "RenderingKernelsLauncher.cuh"
#include "EditKernelsLauncher.cuh"
#include "MonitorKernelsLauncher.cuh"
#include "SimulationResult.cuh"
#include "SelectionResult.cuh"
#include "RenderingData.cuh"

namespace
{
    class CudaInitializer
    {
    public:
        static void init() { [[maybe_unused]] static CudaInitializer instance; }

        CudaInitializer()
        {
            int deviceNumber = getDeviceNumberOfHighestComputeCapability();

            auto result = hipSetDevice(deviceNumber);
            if (result != hipSuccess) {
                throw SystemRequirementNotMetException("CUDA device could not be initialized.");
            }

            std::stringstream stream;
            stream << "device " << deviceNumber << " is set";
            log(Priority::Important, stream.str());
        }

        ~CudaInitializer() { hipDeviceReset(); }

    private:
        int getDeviceNumberOfHighestComputeCapability()
        {
            int result = 0;
            int numberOfDevices;
            CHECK_FOR_CUDA_ERROR(hipGetDeviceCount(&numberOfDevices));
            if (numberOfDevices < 1) {
                throw SystemRequirementNotMetException("No CUDA device found.");
            }
            {
                std::stringstream stream;
                if (1 == numberOfDevices) {
                    stream << "1 CUDA device found";
                } else {
                    stream << numberOfDevices << " CUDA devices found";
                }
                log(Priority::Important, stream.str());
            }

            int highestComputeCapability = 0;
            for (int deviceNumber = 0; deviceNumber < numberOfDevices; ++deviceNumber) {
                hipDeviceProp_t prop;
                CHECK_FOR_CUDA_ERROR(hipGetDeviceProperties(&prop, deviceNumber));

                std::stringstream stream;
                stream << "device " << deviceNumber << ": " << prop.name << " with compute capability " << prop.major
                       << "." << prop.minor;
                log(Priority::Important, stream.str());

                int computeCapability = prop.major * 100 + prop.minor;
                if (computeCapability > highestComputeCapability) {
                    result = deviceNumber;
                    highestComputeCapability = computeCapability;
                }
            }
            if (highestComputeCapability < 600) {
                throw SystemRequirementNotMetException(
                    "No CUDA device with compute capability of 6.0 or higher found.");
            }

            return result;
        }
    };
}

void _CudaSimulationAdapter::initCuda()
{
    CudaInitializer::init();
}

_CudaSimulationAdapter::_CudaSimulationAdapter(uint64_t timestep, Settings const& settings, GpuSettings const& gpuSettings)
{
    CHECK_FOR_CUDA_ERROR(hipGetLastError());

    setSimulationParameters(settings.simulationParameters);
    setSimulationParametersSpots(settings.simulationParametersSpots);
    setGpuConstants(gpuSettings);
    setFlowFieldSettings(settings.flowFieldSettings);

    log(Priority::Important, "initialize simulation");

    _currentTimestep.store(timestep);
    _cudaSimulationData = std::make_shared<SimulationData>();
    _cudaRenderingData = std::make_shared<RenderingData>();
    _cudaSimulationResult = std::make_shared<SimulationResult>();
    _cudaSelectionResult = std::make_shared<SelectionResult>();
    _cudaAccessTO = std::make_shared<DataAccessTO>();
    _cudaMonitorData = std::make_shared<CudaMonitorData>();

    _cudaSimulationData->init({settings.generalSettings.worldSizeX, settings.generalSettings.worldSizeY});
    _cudaRenderingData->init();
    _cudaMonitorData->init();
    _cudaSimulationResult->init();
    _cudaSelectionResult->init();

    _simulationKernels = std::make_shared<_SimulationKernelsLauncher>();
    _dataAccessKernels = std::make_shared<_DataAccessKernelsLauncher>();
    _garbageCollectorKernels = std::make_shared<_GarbageCollectorKernelsLauncher>();
    _renderingKernels = std::make_shared<_RenderingKernelsLauncher>();
    _editKernels = std::make_shared<_EditKernelsLauncher>();
    _monitorKernels = std::make_shared<_MonitorKernelsLauncher>();

    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numStringBytes);
    CudaMemoryManager::getInstance().acquireMemory<char>(Const::MetadataMemorySize, _cudaAccessTO->stringBytes);

    //default array sizes for empty simulation (will be resized later if not sufficient)
    resizeArrays({100000, 100000, 10000});
}

_CudaSimulationAdapter::~_CudaSimulationAdapter()
{
    _cudaSimulationData->free();
    _cudaRenderingData->free();
    _cudaMonitorData->free();
    _cudaSimulationResult->free();
    _cudaSelectionResult->free();

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->tokens);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->stringBytes);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numStringBytes);

    log(Priority::Important, "close simulation");
}

void* _CudaSimulationAdapter::registerImageResource(GLuint image)
{
    hipGraphicsResource* cudaResource;

    CHECK_FOR_CUDA_ERROR(
        hipGraphicsGLRegisterImage(&cudaResource, image, GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly));

    return reinterpret_cast<void*>(cudaResource);
}

void _CudaSimulationAdapter::calcTimestep()
{
    _simulationKernels->calcTimestep(_gpuSettings, _flowFieldSettings, *_cudaSimulationData, *_cudaSimulationResult);
    syncAndCheck();

    automaticResizeArrays();
    ++_currentTimestep;
}

void _CudaSimulationAdapter::drawVectorGraphics(
    float2 const& rectUpperLeft,
    float2 const& rectLowerRight,
    void* cudaResource,
    int2 const& imageSize,
    double zoom)
{
    auto cudaResourceImpl = reinterpret_cast<hipGraphicsResource*>(cudaResource);
    CHECK_FOR_CUDA_ERROR(hipGraphicsMapResources(1, &cudaResourceImpl));

    hipArray* mappedArray;
    CHECK_FOR_CUDA_ERROR(hipGraphicsSubResourceGetMappedArray(&mappedArray, cudaResourceImpl, 0, 0));

    _cudaRenderingData->resizeImageIfNecessary(imageSize);

    _renderingKernels->drawImage(_gpuSettings, rectUpperLeft, rectLowerRight, imageSize, static_cast<float>(zoom), *_cudaSimulationData, *_cudaRenderingData);
    syncAndCheck();

    const size_t widthBytes = sizeof(uint64_t) * imageSize.x;
    CHECK_FOR_CUDA_ERROR(hipMemcpy2DToArray(
        mappedArray,
        0,
        0,
        _cudaRenderingData->imageData,
        widthBytes,
        widthBytes,
        imageSize.y,
        hipMemcpyDeviceToDevice));

    CHECK_FOR_CUDA_ERROR(hipGraphicsUnmapResources(1, &cudaResourceImpl));
}

void _CudaSimulationAdapter::getSimulationData(
    int2 const& rectUpperLeft,
    int2 const& rectLowerRight,
    DataAccessTO const& dataTO)
{
    _dataAccessKernels->getData(_gpuSettings, *_cudaSimulationData, rectUpperLeft, rectLowerRight, *_cudaAccessTO);
    syncAndCheck();

    copyDataTOtoHost(dataTO);
}

void _CudaSimulationAdapter::getSelectedSimulationData(bool includeClusters, DataAccessTO const& dataTO)
{
    _dataAccessKernels->getSelectedData(_gpuSettings, *_cudaSimulationData, includeClusters, *_cudaAccessTO);
    syncAndCheck();

    copyDataTOtoHost(dataTO);
}

void _CudaSimulationAdapter::getInspectedSimulationData(std::vector<uint64_t> entityIds, DataAccessTO const& dataTO)
{
    InspectedEntityIds ids;
    if (entityIds.size() > Const::MaxInspectedEntities) {
        return;
    }
    for (int i = 0; i < entityIds.size(); ++i) {
        ids.values[i] = entityIds.at(i);
    }
    if (entityIds.size() < Const::MaxInspectedEntities) {
        ids.values[entityIds.size()] = 0;
    }
    _dataAccessKernels->getInspectedData(_gpuSettings, *_cudaSimulationData, ids, *_cudaAccessTO);
    syncAndCheck();
    copyDataTOtoHost(dataTO);
}

void _CudaSimulationAdapter::getOverlayData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    _dataAccessKernels->getOverlayData(_gpuSettings, *_cudaSimulationData, rectUpperLeft, rectLowerRight, *_cudaAccessTO);
    syncAndCheck();

    copyToHost(dataTO.numCells, _cudaAccessTO->numCells);
    copyToHost(dataTO.numParticles, _cudaAccessTO->numParticles);
    copyToHost(dataTO.cells, _cudaAccessTO->cells, *dataTO.numCells);
    copyToHost(dataTO.particles, _cudaAccessTO->particles, *dataTO.numParticles);
}

void _CudaSimulationAdapter::addAndSelectSimulationData(DataAccessTO const& dataTO)
{
    copyDataTOtoDevice(dataTO);
    _editKernels->removeSelection(_gpuSettings, *_cudaSimulationData);
    _dataAccessKernels->addData(_gpuSettings, *_cudaSimulationData, *_cudaAccessTO, true, true);
    syncAndCheck();
}

void _CudaSimulationAdapter::setSimulationData(DataAccessTO const& dataTO)
{
    copyDataTOtoDevice(dataTO);
    _dataAccessKernels->clearData(_gpuSettings, *_cudaSimulationData);
    _dataAccessKernels->addData(_gpuSettings, *_cudaSimulationData, *_cudaAccessTO, false, false);
    syncAndCheck();
}

void _CudaSimulationAdapter::removeSelectedEntities(bool includeClusters)
{
    _editKernels->removeSelectedEntities(_gpuSettings, *_cudaSimulationData, includeClusters);
    syncAndCheck();
}

void _CudaSimulationAdapter::changeInspectedSimulationData(DataAccessTO const& changeDataTO)
{
    copyDataTOtoDevice(changeDataTO);
    _editKernels->changeSimulationData(_gpuSettings, *_cudaSimulationData, *_cudaAccessTO);
    syncAndCheck();
}

void _CudaSimulationAdapter::applyForce(ApplyForceData const& applyData)
{
    _editKernels->applyForce(_gpuSettings, *_cudaSimulationData, applyData);
    syncAndCheck();
}

void _CudaSimulationAdapter::switchSelection(PointSelectionData const& pointData)
{
    _editKernels->switchSelection(_gpuSettings, *_cudaSimulationData, pointData);
    syncAndCheck();
}

void _CudaSimulationAdapter::swapSelection(PointSelectionData const& pointData)
{
    _editKernels->swapSelection(_gpuSettings, *_cudaSimulationData, pointData);
    syncAndCheck();
}

void _CudaSimulationAdapter::setSelection(AreaSelectionData const& selectionData)
{
    _editKernels->setSelection(_gpuSettings, *_cudaSimulationData, selectionData);
}

 SelectionShallowData _CudaSimulationAdapter::getSelectionShallowData()
{
    _editKernels->getSelectionShallowData(_gpuSettings, *_cudaSimulationData, *_cudaSelectionResult);
    syncAndCheck();
    return _cudaSelectionResult->getSelectionShallowData();
}

void _CudaSimulationAdapter::shallowUpdateSelectedEntities(ShallowUpdateSelectionData const& shallowUpdateData)
{
    _editKernels->shallowUpdateSelectedEntities(_gpuSettings, *_cudaSimulationData, shallowUpdateData);
    syncAndCheck();
}

void _CudaSimulationAdapter::removeSelection()
{
    _editKernels->removeSelection(_gpuSettings, *_cudaSimulationData);
    syncAndCheck();
}

void _CudaSimulationAdapter::updateSelection()
{
    _editKernels->updateSelection(_gpuSettings, *_cudaSimulationData);
    syncAndCheck();
}

void _CudaSimulationAdapter::colorSelectedEntities(unsigned char color, bool includeClusters)
{
    _editKernels->colorSelectedCells(_gpuSettings, *_cudaSimulationData, color, includeClusters);
    syncAndCheck();
}

void _CudaSimulationAdapter::reconnectSelectedEntities()
{
    _editKernels->reconnectSelectedEntities(_gpuSettings, *_cudaSimulationData);
    syncAndCheck();
}

void _CudaSimulationAdapter::setGpuConstants(GpuSettings const& gpuConstants)
{
    _gpuSettings = gpuConstants;

    CHECK_FOR_CUDA_ERROR(
        hipMemcpyToSymbol(HIP_SYMBOL(cudaThreadSettings), &gpuConstants, sizeof(GpuSettings), 0, hipMemcpyHostToDevice));
}

auto _CudaSimulationAdapter::getArraySizes() const -> ArraySizes
{
    return {
        _cudaSimulationData->entities.cells.getSize_host(),
        _cudaSimulationData->entities.particles.getSize_host(),
        _cudaSimulationData->entities.tokens.getSize_host()};
}

OverallStatistics _CudaSimulationAdapter::getMonitorData()
{
    _monitorKernels->getMonitorData(_gpuSettings, *_cudaSimulationData, *_cudaMonitorData);
    syncAndCheck();
    
    OverallStatistics result;
    auto monitorData = _cudaMonitorData->getMonitorData(getCurrentTimestep());
    result.timeStep = monitorData.timeStep;
    result.numCells = monitorData.numCells;
    result.numParticles = monitorData.numParticles;
    result.numTokens = monitorData.numTokens;
    result.totalInternalEnergy = monitorData.totalInternalEnergy;

    auto processStatistics = _cudaSimulationResult->getStatistics();
    result.numCreatedCells = processStatistics.createdCells;
    result.numSuccessfulAttacks = processStatistics.sucessfulAttacks;
    result.numFailedAttacks = processStatistics.failedAttacks;
    result.numMuscleActivities = processStatistics.muscleActivities;
    return result;
}

uint64_t _CudaSimulationAdapter::getCurrentTimestep() const
{
    return _currentTimestep.load();
}

void _CudaSimulationAdapter::setCurrentTimestep(uint64_t timestep)
{
    _currentTimestep.store(timestep);
}

void _CudaSimulationAdapter::setSimulationParameters(SimulationParameters const& parameters)
{
    CHECK_FOR_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &parameters, sizeof(SimulationParameters), 0, hipMemcpyHostToDevice));
}

void _CudaSimulationAdapter::setSimulationParametersSpots(SimulationParametersSpots const& spots)
{
    CHECK_FOR_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(
        cudaSimulationParametersSpots), &spots, sizeof(SimulationParametersSpots), 0, hipMemcpyHostToDevice));
}

void _CudaSimulationAdapter::setFlowFieldSettings(FlowFieldSettings const& settings)
{
    CHECK_FOR_CUDA_ERROR(
        hipMemcpyToSymbol(HIP_SYMBOL(cudaFlowFieldSettings), &settings, sizeof(FlowFieldSettings), 0, hipMemcpyHostToDevice));

    _flowFieldSettings = settings;
}


void _CudaSimulationAdapter::clear()
{
    _dataAccessKernels->clearData(_gpuSettings, *_cudaSimulationData);
    syncAndCheck();
}

void _CudaSimulationAdapter::resizeArraysIfNecessary(ArraySizes const& additionals)
{
    if (_cudaSimulationData->shouldResize(
            additionals.cellArraySize, additionals.particleArraySize, additionals.tokenArraySize)) {
        resizeArrays(additionals);
    }
}

void _CudaSimulationAdapter::syncAndCheck()
{
    hipDeviceSynchronize();
    CHECK_FOR_CUDA_ERROR(hipGetLastError());
}

void _CudaSimulationAdapter::copyDataTOtoDevice(DataAccessTO const& dataTO)
{
    copyToDevice(_cudaAccessTO->numCells, dataTO.numCells);
    copyToDevice(_cudaAccessTO->numParticles, dataTO.numParticles);
    copyToDevice(_cudaAccessTO->numTokens, dataTO.numTokens);
    copyToDevice(_cudaAccessTO->numStringBytes, dataTO.numStringBytes);

    copyToDevice(_cudaAccessTO->cells, dataTO.cells, *dataTO.numCells);
    copyToDevice(_cudaAccessTO->particles, dataTO.particles, *dataTO.numParticles);
    copyToDevice(_cudaAccessTO->tokens, dataTO.tokens, *dataTO.numTokens);
    copyToDevice(_cudaAccessTO->stringBytes, dataTO.stringBytes, *dataTO.numStringBytes);
}

void _CudaSimulationAdapter::copyDataTOtoHost(DataAccessTO const& dataTO)
{
    copyToHost(dataTO.numCells, _cudaAccessTO->numCells);
    copyToHost(dataTO.numParticles, _cudaAccessTO->numParticles);
    copyToHost(dataTO.numTokens, _cudaAccessTO->numTokens);
    copyToHost(dataTO.numStringBytes, _cudaAccessTO->numStringBytes);

    copyToHost(dataTO.cells, _cudaAccessTO->cells, *dataTO.numCells);
    copyToHost(dataTO.particles, _cudaAccessTO->particles, *dataTO.numParticles);
    copyToHost(dataTO.tokens, _cudaAccessTO->tokens, *dataTO.numTokens);
    copyToHost(dataTO.stringBytes, _cudaAccessTO->stringBytes, *dataTO.numStringBytes);
}

void _CudaSimulationAdapter::automaticResizeArrays()
{
    //make check after every 10th time step
    if (_currentTimestep.load() % 10 == 0) {
        if (_cudaSimulationResult->isArrayResizeNeeded()) {
            resizeArrays({0, 0, 0});
        }
    }
}

void _CudaSimulationAdapter::resizeArrays(ArraySizes const& additionals)
{
    log(Priority::Important, "resize arrays");

    _cudaSimulationData->resizeEntitiesForCleanup(
        additionals.cellArraySize, additionals.particleArraySize, additionals.tokenArraySize);
    if (!_cudaSimulationData->isEmpty()) {
        _garbageCollectorKernels->copyArrays(_gpuSettings, *_cudaSimulationData);
        syncAndCheck();

        _cudaSimulationData->resizeRemainings();
        _cudaSimulationData->swap();
    } else {
        _cudaSimulationData->resizeRemainings();
    }

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->tokens);

    auto cellArraySize = _cudaSimulationData->entities.cells.getSize_host();
    auto tokenArraySize = _cudaSimulationData->entities.tokens.getSize_host();
    CudaMemoryManager::getInstance().acquireMemory<CellAccessTO>(cellArraySize, _cudaAccessTO->cells);
    CudaMemoryManager::getInstance().acquireMemory<ParticleAccessTO>(cellArraySize, _cudaAccessTO->particles);
    CudaMemoryManager::getInstance().acquireMemory<TokenAccessTO>(tokenArraySize, _cudaAccessTO->tokens);

    CHECK_FOR_CUDA_ERROR(hipGetLastError());

    log(Priority::Unimportant, "cell array size: " + std::to_string(cellArraySize));
    log(Priority::Unimportant, "particle array size: " + std::to_string(cellArraySize));
    log(Priority::Unimportant, "token array size: " + std::to_string(tokenArraySize));

        auto const memorySizeAfter = CudaMemoryManager::getInstance().getSizeOfAcquiredMemory();
    log(Priority::Important, std::to_string(memorySizeAfter / (1024 * 1024)) + " MB GPU memory acquired");
}
