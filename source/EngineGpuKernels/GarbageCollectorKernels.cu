#include "hip/hip_runtime.h"
﻿#include "GarbageCollectorKernels.cuh"

__global__ void cudaPreparePointerArraysForCleanup(SimulationData data)
{
    data.entitiesForCleanup.particlePointers.reset();
    data.entitiesForCleanup.cellPointers.reset();
    data.entitiesForCleanup.tokenPointers.reset();
}

__global__ void cudaPrepareArraysForCleanup(SimulationData data)
{
    data.entitiesForCleanup.particles.reset();
    data.entitiesForCleanup.cells.reset();
    data.entitiesForCleanup.tokens.reset();
}

__global__ void cudaCleanupCellsStep1(Array<Cell*> cellPointers, Array<Cell> cells)
{
    //assumes that cellPointers are already cleaned up
    PartitionData pointerBlock = calcPartition(cellPointers.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    int numCellsToCopy = pointerBlock.numElements();
    if (numCellsToCopy > 0) {
        auto newCells = cells.getNewSubarray(numCellsToCopy);

        int newCellIndex = 0;
        for (int index = pointerBlock.startIndex; index <= pointerBlock.endIndex; ++index) {
            auto& cellPointer = cellPointers.at(index);
            auto& newCell = newCells[newCellIndex];
            newCell = *cellPointer;

            cellPointer->tag = &newCell - cells.getArray();  //save index of new cell in old cell
            cellPointer = &newCell;

            ++newCellIndex;
        }
    }
}

__global__ void cudaCleanupCellsStep2(Array<Token*> tokenPointers, Array<Cell> cells)
{
    {
        auto partition = calcPartition(cells.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& cell = cells.at(index);
            for (int i = 0; i < cell.numConnections; ++i) {
                auto& connectedCell = cell.connections[i].cell;
                cell.connections[i].cell = &cells.at(connectedCell->tag);
            }
        }
    }
    {
        auto partition = calcPartition(tokenPointers.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            if (auto& token = tokenPointers.at(index)) {
                token->cell = &cells.at(token->cell->tag);
                token->sourceCell = &cells.at(token->sourceCell->tag);
            }
        }
    }
}

__global__ void cudaCleanupTokens(Array<Token*> tokenPointers, Array<Token> newToken)
{
    auto partition = calcPartition(tokenPointers.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    if (partition.numElements() > 0) {
        Token* newEntities = newToken.getNewSubarray(partition.numElements());

        int targetIndex = 0;
        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& token = tokenPointers.at(index);
            newEntities[targetIndex] = *token;
            token = &newEntities[targetIndex];
            ++targetIndex;
        }
    }
}

__global__ void cudaCleanupCellMap(SimulationData data)
{
    data.cellMap.cleanup_system();
}

__global__ void cudaCleanupParticleMap(SimulationData data)
{
    data.particleMap.cleanup_system();
}

__global__ void cudaSwapPointerArrays(SimulationData data)
{
    data.entities.particlePointers.swapContent(data.entitiesForCleanup.particlePointers);
    data.entities.cellPointers.swapContent(data.entitiesForCleanup.cellPointers);
    data.entities.tokenPointers.swapContent(data.entitiesForCleanup.tokenPointers);
}

__global__ void cudaSwapArrays(SimulationData data)
{
    data.entities.cells.swapContent(data.entitiesForCleanup.cells);
    data.entities.tokens.swapContent(data.entitiesForCleanup.tokens);
    data.entities.particles.swapContent(data.entitiesForCleanup.particles);
}


__global__ void cudaCleanupParticles(Array<Particle*> particlePointers, Array<Particle> particles)
{
    //assumes that particlePointers are already cleaned up
    PartitionData pointerBlock = calcPartition(particlePointers.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    int numParticlesToCopy = pointerBlock.numElements();
    if (numParticlesToCopy > 0) {
        auto newParticles = particles.getNewSubarray(numParticlesToCopy);

        int newParticleIndex = 0;
        for (int index = pointerBlock.startIndex; index <= pointerBlock.endIndex; ++index) {
            auto& particlePointer = particlePointers.at(index);
            auto& newParticle = newParticles[newParticleIndex];
            newParticle = *particlePointer;
            particlePointer = &newParticle;

            ++newParticleIndex;
        }
    }
}

__global__ void cudaCheckIfCleanupIsNecessary(SimulationData data, bool* result)
{
    if (data.entities.particles.getNumEntries() > data.entities.particles.getSize() * Const::ArrayFillLevelFactor
        || data.entities.cells.getNumEntries() > data.entities.cells.getSize() * Const::ArrayFillLevelFactor
        || data.entities.tokens.getNumEntries() > data.entities.tokens.getSize() * Const::ArrayFillLevelFactor) {
        *result = true;
    } else {
        *result = false;
    }
}

/*
__global__ void cleanupMetadata(Array<Cluster*> clusterPointers, DynamicMemory strings)
{
    auto const clusterBlock = calcPartition(clusterPointers.getNumEntries(), blockIdx.x, gridDim.x);
    for (int clusterIndex = clusterBlock.startIndex; clusterIndex <= clusterBlock.endIndex; ++clusterIndex) {
        auto& cluster = clusterPointers.at(clusterIndex);

        if (0 == threadIdx.x) {
            auto const len = cluster->metadata.nameLen;
            auto newName = strings.getArray<char>(len);
            for(int i = 0; i < len; ++i) {
                newName[i] = cluster->metadata.name[i];
            }
            cluster->metadata.name = newName;
        }

        auto const cellBlock = calcPartition(cluster->numCellPointers, threadIdx.x, blockDim.x);
        for (int cellIndex = cellBlock.startIndex; cellIndex <= cellBlock.endIndex; ++cellIndex) {
            auto& cell = cluster->cellPointers[cellIndex];
            {
                auto const len = cell->metadata.nameLen;
                auto newName = strings.getArray<char>(len);
                for (int i = 0; i < len; ++i) {
                    newName[i] = cell->metadata.name[i];
                }
                cell->metadata.name = newName;
            }
            {
                auto const len = cell->metadata.descriptionLen;
                auto newDescription = strings.getArray<char>(len);
                for (int i = 0; i < len; ++i) {
                    newDescription[i] = cell->metadata.description[i];
                }
                cell->metadata.description = newDescription;
            }
            {
                auto const len = cell->metadata.sourceCodeLen;
                auto newSourceCode = strings.getArray<char>(len);
                for (int i = 0; i < len; ++i) {
                    newSourceCode[i] = cell->metadata.sourceCode[i];
                }
                cell->metadata.sourceCode = newSourceCode;
            }
        }
    }
}
*/
