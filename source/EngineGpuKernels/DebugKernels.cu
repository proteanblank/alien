#include "hip/hip_runtime.h"
﻿#include "DebugKernels.cuh"

__device__ void DEBUG_checkCells(SimulationData& data, float* sumEnergy, int location)
{
    auto& cells = data.entities.cellPointers;
    auto partition = calcPartition(cells.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        if (auto& cell = cells.at(index)) {

            for (int i = 0; i < cell->numConnections; ++i) {
                auto connectingCell = cell->connections[i].cell;

                auto displacement = connectingCell->absPos - cell->absPos;
                data.cellMap.mapDisplacementCorrection(displacement);
                auto actualDistance = Math::length(displacement);
                if (actualDistance > 14) {
                    printf("distance too large at %d\n", location);
                    CUDA_THROW_NOT_IMPLEMENTED();
                }
            }
            if (cell->energy < 0 || isnan(cell->energy)) {
                printf("cell energy invalid at %d", location);
                CUDA_THROW_NOT_IMPLEMENTED();
            }
            atomicAdd(sumEnergy, cell->energy);
        }
    }
}

__device__ void DEBUG_checkParticles(SimulationData& data, float* sumEnergy, int location)
{
    auto partition = calcPartition(data.entities.particlePointers.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    for (int particleIndex = partition.startIndex; particleIndex <= partition.endIndex; ++particleIndex) {
        if (auto& particle = data.entities.particlePointers.at(particleIndex)) {
            if (particle->energy < 0 || isnan(particle->energy)) {
                printf("particle energy invalid at %d", location);
                CUDA_THROW_NOT_IMPLEMENTED();
            }
            atomicAdd(sumEnergy, particle->energy);
        }
    }
}

__global__ void DEBUG_checkCellsAndParticles(SimulationData data, float* sumEnergy, int location)
{
    DEBUG_checkCells(data, sumEnergy, location);
    DEBUG_checkParticles(data, sumEnergy, location);
}

/*
__global__ void DEBUG_kernel(SimulationData data, int location)
{
    float* sumEnergy = new float;
    *sumEnergy = 0;

    DEPRECATED_KERNEL_CALL_SYNC(DEBUG_checkCellsAndParticles, data, sumEnergy, location);

    float const expectedEnergy = 187500;
    if (abs(*sumEnergy - expectedEnergy) > 1) {
        printf("location: %d, actual energy: %f, expected energy: %f\n", location, *sumEnergy, expectedEnergy);
        CUDA_THROW_NOT_IMPLEMENTED();
    }
    delete sumEnergy;
}
*/
