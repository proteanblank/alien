﻿#include "Entities.cuh"

#include "Cell.cuh"
#include "Token.cuh"
#include "Particle.cuh"

void Entities::init()
{
    cellPointers.init();
    cells.init();
    tokenPointers.init();
    tokens.init();
    particles.init();
    particlePointers.init();
    dynamicMemory.init();
    dynamicMemory.resize(Const::MetadataMemorySize);
}

void Entities::free()
{
    cellPointers.free();
    cells.free();
    tokenPointers.free();
    tokens.free();
    particles.free();
    particlePointers.free();
    dynamicMemory.free();
}
