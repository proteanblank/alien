#include "hip/hip_runtime.h"
﻿#include "DataAccessKernels.cuh"

namespace
{
    __device__ void copyString(int& targetLen, int& targetStringIndex, int sourceLen, char* sourceString, int& numStringBytes, char*& stringBytes)
    {
        targetLen = sourceLen;
        if (sourceLen > 0) {
            targetStringIndex = atomicAdd(&numStringBytes, sourceLen);
            for (int i = 0; i < sourceLen; ++i) {
                stringBytes[targetStringIndex + i] = sourceString[i];
            }
        }
    }

    __device__ void createCellTO(Cell* cell, DataAccessTO& dataTO, Cell* cellArrayStart)
    {
        auto cellTOIndex = atomicAdd(dataTO.numCells, 1);
        auto& cellTO = dataTO.cells[cellTOIndex];

        cellTO.id = cell->id;
        cellTO.pos = cell->absPos;
        cellTO.vel = cell->vel;
        cellTO.energy = cell->energy;
        cellTO.maxConnections = cell->maxConnections;
        cellTO.numConnections = cell->numConnections;
        cellTO.branchNumber = cell->branchNumber;
        cellTO.tokenBlocked = cell->tokenBlocked;
        cellTO.cellFunctionType = cell->cellFunctionType;
        cellTO.numStaticBytes = cell->numStaticBytes;
        cellTO.tokenUsages = cell->tokenUsages;
        cellTO.metadata.color = cell->metadata.color;

        copyString(
            cellTO.metadata.nameLen, cellTO.metadata.nameStringIndex, cell->metadata.nameLen, cell->metadata.name, *dataTO.numStringBytes, dataTO.stringBytes);
        copyString(
            cellTO.metadata.descriptionLen,
            cellTO.metadata.descriptionStringIndex,
            cell->metadata.descriptionLen,
            cell->metadata.description,
            *dataTO.numStringBytes,
            dataTO.stringBytes);
        copyString(
            cellTO.metadata.sourceCodeLen,
            cellTO.metadata.sourceCodeStringIndex,
            cell->metadata.sourceCodeLen,
            cell->metadata.sourceCode,
            *dataTO.numStringBytes,
            dataTO.stringBytes);

        cell->tag = cellTOIndex;
        for (int i = 0; i < cell->numConnections; ++i) {
            auto connectingCell = cell->connections[i].cell;
            cellTO.connections[i].cellIndex = connectingCell - cellArrayStart;
            cellTO.connections[i].distance = cell->connections[i].distance;
            cellTO.connections[i].angleFromPrevious = cell->connections[i].angleFromPrevious;
        }
        for (int i = 0; i < MAX_CELL_STATIC_BYTES; ++i) {
            cellTO.staticData[i] = cell->staticData[i];
        }
        cellTO.numMutableBytes = cell->numMutableBytes;
        for (int i = 0; i < MAX_CELL_MUTABLE_BYTES; ++i) {
            cellTO.mutableData[i] = cell->mutableData[i];
        }
    }

    __device__ void createParticleTO(Particle* particle, DataAccessTO& dataTO)
    {
        int particleTOIndex = atomicAdd(dataTO.numParticles, 1);
        ParticleAccessTO& particleTO = dataTO.particles[particleTOIndex];

        particleTO.id = particle->id;
        particleTO.pos = particle->absPos;
        particleTO.vel = particle->vel;
        particleTO.energy = particle->energy;
    }

}

/************************************************************************/
/* Main                                                                 */
/************************************************************************/
__global__ void cudaGetSelectedCellDataWithoutConnections(SimulationData data, bool includeClusters, DataAccessTO dataTO)
{
    auto const& cells = data.entities.cellPointers;
    auto const partition = calcAllThreadsPartition(cells.getNumEntries());
    auto const cellArrayStart = data.entities.cells.getArray();

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        auto& cell = cells.at(index);
        if ((includeClusters && cell->selected == 0) || (!includeClusters && cell->selected != 1)) {
            cell->tag = -1;
            continue;
        }
        createCellTO(cell, dataTO, cellArrayStart);
    }
}

__global__ void cudaGetSelectedParticleData(SimulationData data, DataAccessTO access)
{
    PartitionData particleBlock = calcPartition(data.entities.particlePointers.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    for (int particleIndex = particleBlock.startIndex; particleIndex <= particleBlock.endIndex; ++particleIndex) {
        auto const& particle = data.entities.particlePointers.at(particleIndex);
        if (particle->selected == 0) {
            continue;
        }

        createParticleTO(particle, access);
    }
}

__global__ void cudaGetInspectedCellDataWithoutConnections(InspectedEntityIds ids, SimulationData data, DataAccessTO dataTO)
{
    auto const& cells = data.entities.cellPointers;
    auto const partition = calcAllThreadsPartition(cells.getNumEntries());
    auto const cellArrayStart = data.entities.cells.getArray();

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        auto& cell = cells.at(index);

        bool found = false;
        for (int i = 0; i < Const::MaxInspectedEntities; ++i) {
            if (ids.values[i] == 0) {
                break;
            }
            if (ids.values[i] == cell->id) {
                found = true;
            }
        }
        if (!found) {
            cell->tag = -1;
            continue;
        }

        createCellTO(cell, dataTO, cellArrayStart);
    }
}

__global__ void cudaGetInspectedParticleData(InspectedEntityIds ids, SimulationData data, DataAccessTO access)
{
    PartitionData particleBlock = calcAllThreadsPartition(data.entities.particlePointers.getNumEntries());

    for (int particleIndex = particleBlock.startIndex; particleIndex <= particleBlock.endIndex; ++particleIndex) {
        auto const& particle = data.entities.particlePointers.at(particleIndex);
        bool found = false;
        for (int i = 0; i < Const::MaxInspectedEntities; ++i) {
            if (ids.values[i] == 0) {
                break;
            }
            if (ids.values[i] == particle->id) {
                found = true;
            }
        }
        if (!found) {
            continue;
        }

        createParticleTO(particle, access);
    }
}

__global__ void cudaGetOverlayData(int2 rectUpperLeft, int2 rectLowerRight, SimulationData data, DataAccessTO dataTO)
{
    {
        auto const& cells = data.entities.cellPointers;
        auto const partition = calcAllThreadsPartition(cells.getNumEntries());

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& cell = cells.at(index);

            auto pos = cell->absPos;
            data.cellMap.mapPosCorrection(pos);
            if (!isContainedInRect(rectUpperLeft, rectLowerRight, pos)) {
                continue;
            }
            auto cellTOIndex = atomicAdd(dataTO.numCells, 1);
            auto& cellTO = dataTO.cells[cellTOIndex];

            cellTO.id = cell->id;
            cellTO.pos = cell->absPos;
            cellTO.cellFunctionType = cell->cellFunctionType;
            cellTO.selected = cell->selected;
            cellTO.branchNumber = cell->branchNumber;
        }
    }
    {
        auto const& particles = data.entities.particlePointers;
        auto const partition = calcAllThreadsPartition(particles.getNumEntries());

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& particle = particles.at(index);

            auto pos = particle->absPos;
            data.particleMap.mapPosCorrection(pos);
            if (!isContainedInRect(rectUpperLeft, rectLowerRight, pos)) {
                continue;
            }
            auto particleTOIndex = atomicAdd(dataTO.numParticles, 1);
            auto& particleTO = dataTO.particles[particleTOIndex];

            particleTO.id = particle->id;
            particleTO.pos = particle->absPos;
            particleTO.selected = particle->selected;
        }
    }
}

//tags cell with cellTO index and tags cellTO connections with cell index
__global__ void cudaGetCellDataWithoutConnections(int2 rectUpperLeft, int2 rectLowerRight, SimulationData data, DataAccessTO dataTO)
{
    auto const& cells = data.entities.cellPointers;
    auto const partition = calcAllThreadsPartition(cells.getNumEntries());
    auto const cellArrayStart = data.entities.cells.getArray();

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        auto& cell = cells.at(index);

        auto pos = cell->absPos;
        data.cellMap.mapPosCorrection(pos);
        if (!isContainedInRect(rectUpperLeft, rectLowerRight, pos)) {
            cell->tag = -1;
            continue;
        }

        createCellTO(cell, dataTO, cellArrayStart);
    }
}

__global__ void cudaResolveConnections(SimulationData data, DataAccessTO dataTO)
{
    auto const partition = calcAllThreadsPartition(*dataTO.numCells);
    auto const firstCell = data.entities.cells.getArray();

    for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
        auto& cellTO = dataTO.cells[index];

        for (int i = 0; i < cellTO.numConnections; ++i) {
            auto const cellIndex = cellTO.connections[i].cellIndex;
            cellTO.connections[i].cellIndex = data.entities.cells.at(cellIndex).tag;
        }
    }
}

__global__ void cudaGetTokenData(SimulationData data, DataAccessTO dataTO)
{
    auto const& tokens = data.entities.tokenPointers;

    auto partition = calcPartition(tokens.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);
    for (auto tokenIndex = partition.startIndex; tokenIndex <= partition.endIndex; ++tokenIndex) {
        auto token = tokens.at(tokenIndex);

        if (token->cell->tag == -1) {
            continue;
        }

        auto tokenTOIndex = atomicAdd(dataTO.numTokens, 1);
        auto& tokenTO = dataTO.tokens[tokenTOIndex];

        tokenTO.energy = token->energy;
        for (int i = 0; i < cudaSimulationParameters.tokenMemorySize; ++i) {
            tokenTO.memory[i] = token->memory[i];
        }
        tokenTO.cellIndex = token->cell->tag;
        tokenTO.sequenceNumber = tokenIndex;
    }
}

__global__ void cudaGetParticleData(int2 rectUpperLeft, int2 rectLowerRight, SimulationData data, DataAccessTO access)
{
    PartitionData particleBlock = calcPartition(data.entities.particlePointers.getNumEntries(), threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

    for (int particleIndex = particleBlock.startIndex; particleIndex <= particleBlock.endIndex; ++particleIndex) {
        auto const& particle = data.entities.particlePointers.at(particleIndex);
        auto pos = particle->absPos;
        data.particleMap.mapPosCorrection(pos);
        if (!isContainedInRect(rectUpperLeft, rectLowerRight, pos)) {
            continue;
        }

        createParticleTO(particle, access);
    }
}

__global__ void cudaCreateDataFromTO(SimulationData data, DataAccessTO dataTO, bool selectNewData, bool createIds)
{
    __shared__ EntityFactory factory;
    if (0 == threadIdx.x) {
        factory.init(&data);
    }
    __syncthreads();

    auto particlePartition = calcPartition(*dataTO.numParticles, threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);
    for (int index = particlePartition.startIndex; index <= particlePartition.endIndex; ++index) {
        auto particle = factory.createParticleFromTO(dataTO.particles[index], createIds);
        if (selectNewData) {
            particle->selected = 1;
        }
    }

    auto cellPartition = calcPartition(*dataTO.numCells, threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);
    auto cellTargetArray = data.entities.cells.getArray() + data.entities.cells.getNumOrigEntries();
    for (int index = cellPartition.startIndex; index <= cellPartition.endIndex; ++index) {
        auto cell = factory.createCellFromTO(index, dataTO.cells[index], cellTargetArray, &dataTO, createIds);
        if (selectNewData) {
            cell->selected = 1;
        }
    }

    auto tokenPartition = calcPartition(*dataTO.numTokens, threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);
    for (int index = tokenPartition.startIndex; index <= tokenPartition.endIndex; ++index) {
        factory.createTokenFromTO(dataTO.tokens[index], cellTargetArray);
    }
}

__global__ void cudaAdaptNumberGenerator(CudaNumberGenerator numberGen, DataAccessTO dataTO)
{
    {
        auto const partition = calcPartition(*dataTO.numCells, threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto const& cell = dataTO.cells[index];
            numberGen.adaptMaxId(cell.id);
        }
    }
    {
        auto const partition = calcPartition(*dataTO.numParticles, threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto const& particle = dataTO.particles[index];
            numberGen.adaptMaxId(particle.id);
        }
    }
}

__global__ void cudaClearDataTO(DataAccessTO dataTO)
{
    *dataTO.numCells = 0;
    *dataTO.numParticles = 0;
    *dataTO.numTokens = 0;
    *dataTO.numStringBytes = 0;
}

__global__ void cudaClearData(SimulationData data)
{
    data.entities.cellPointers.reset();
    data.entities.tokenPointers.reset();
    data.entities.particlePointers.reset();
    data.entities.cells.reset();
    data.entities.tokens.reset();
    data.entities.particles.reset();
    data.entities.dynamicMemory.reset();
}

__global__ void cudaSaveNumEntries(SimulationData data)
{
    data.entities.saveNumEntries();
}
