﻿#include "GarbageCollectorKernelsLauncher.cuh"

_GarbageCollectorKernelsLauncher::_GarbageCollectorKernelsLauncher()
{
    CudaMemoryManager::getInstance().acquireMemory<bool>(1, _cudaBool);
}

_GarbageCollectorKernelsLauncher::~_GarbageCollectorKernelsLauncher()
{
    CudaMemoryManager::getInstance().freeMemory(_cudaBool);
}

void _GarbageCollectorKernelsLauncher::cleanupAfterTimestep(GpuSettings const& gpuSettings, SimulationData const& data)
{
    KERNEL_CALL(cudaCleanupCellMap, data);
    KERNEL_CALL(cudaCleanupParticleMap, data);

    KERNEL_CALL_1_1(cudaPreparePointerArraysForCleanup, data);
    KERNEL_CALL(cudaCleanupPointerArray<Particle*>, data.entities.particlePointers, data.entitiesForCleanup.particlePointers);
    KERNEL_CALL(cudaCleanupPointerArray<Cell*>, data.entities.cellPointers, data.entitiesForCleanup.cellPointers);
    KERNEL_CALL(cudaCleanupPointerArray<Token*>, data.entities.tokenPointers, data.entitiesForCleanup.tokenPointers);
    KERNEL_CALL_1_1(cudaSwapPointerArrays, data);
    hipDeviceSynchronize();

    KERNEL_CALL_1_1(cudaCheckIfCleanupIsNecessary, data, _cudaBool);
    hipDeviceSynchronize();
    if (copyToHost(_cudaBool)) {
        KERNEL_CALL_1_1(cudaPrepareArraysForCleanup, data);
        KERNEL_CALL(cudaCleanupParticles, data.entities.particlePointers, data.entitiesForCleanup.particles);
        KERNEL_CALL(cudaCleanupCellsStep1, data.entities.cellPointers, data.entitiesForCleanup.cells);
        KERNEL_CALL(cudaCleanupCellsStep2, data.entities.tokenPointers, data.entitiesForCleanup.cells);
        KERNEL_CALL(cudaCleanupTokens, data.entities.tokenPointers, data.entitiesForCleanup.tokens);
        KERNEL_CALL_1_1(cudaSwapArrays, data);
    }
}

void _GarbageCollectorKernelsLauncher::cleanupAfterDataManipulation(GpuSettings const& gpuSettings, SimulationData const& data)
{
    KERNEL_CALL_1_1(cudaPreparePointerArraysForCleanup, data);
    KERNEL_CALL(cudaCleanupPointerArray<Particle*>, data.entities.particlePointers, data.entitiesForCleanup.particlePointers);
    KERNEL_CALL(cudaCleanupPointerArray<Cell*>, data.entities.cellPointers, data.entitiesForCleanup.cellPointers);
    KERNEL_CALL(cudaCleanupPointerArray<Token*>, data.entities.tokenPointers, data.entitiesForCleanup.tokenPointers);
    KERNEL_CALL_1_1(cudaSwapPointerArrays, data);

    KERNEL_CALL_1_1(cudaPrepareArraysForCleanup, data);
    KERNEL_CALL(cudaCleanupParticles, data.entities.particlePointers, data.entitiesForCleanup.particles);
    KERNEL_CALL(cudaCleanupCellsStep1, data.entities.cellPointers, data.entitiesForCleanup.cells);
    KERNEL_CALL(cudaCleanupCellsStep2, data.entities.tokenPointers, data.entitiesForCleanup.cells);
    KERNEL_CALL(cudaCleanupTokens, data.entities.tokenPointers, data.entitiesForCleanup.tokens);
    KERNEL_CALL_1_1(cudaSwapArrays, data);
}

void _GarbageCollectorKernelsLauncher::copyArrays(GpuSettings const& gpuSettings, SimulationData const& simulationData)
{
    KERNEL_CALL_1_1(cudaPreparePointerArraysForCleanup, simulationData);
    KERNEL_CALL(cudaCleanupPointerArray<Particle*>, simulationData.entities.particlePointers, simulationData.entitiesForCleanup.particlePointers);
    KERNEL_CALL(cudaCleanupPointerArray<Cell*>, simulationData.entities.cellPointers, simulationData.entitiesForCleanup.cellPointers);
    KERNEL_CALL(cudaCleanupPointerArray<Token*>, simulationData.entities.tokenPointers, simulationData.entitiesForCleanup.tokenPointers);

    KERNEL_CALL_1_1(cudaPrepareArraysForCleanup, simulationData);
    KERNEL_CALL(cudaCleanupParticles, simulationData.entitiesForCleanup.particlePointers, simulationData.entitiesForCleanup.particles);
    KERNEL_CALL(cudaCleanupCellsStep1, simulationData.entitiesForCleanup.cellPointers, simulationData.entitiesForCleanup.cells);
    KERNEL_CALL(cudaCleanupCellsStep2, simulationData.entitiesForCleanup.tokenPointers, simulationData.entitiesForCleanup.cells);
    KERNEL_CALL(cudaCleanupTokens, simulationData.entitiesForCleanup.tokenPointers, simulationData.entitiesForCleanup.tokens);
}
